#include "hip/hip_runtime.h"
/*
 * otbCudaFCLSFilter.cu
 *
 */

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_vector_types.h>

//Block thread size
#define BLOCK_SIZE (16*16)

//#define num_bands 224
//#define num_endmembers 5

#define CUDA_SAFE_CALL( call ) \
  do \
  { \
  hipError_t status = call; \
  check_error(status); \
  } while(0)

void check_error(hipError_t error)
{

#define ELSEIF_CUDA_HANDLE_ERROR_CODE( code ) \
  else if(error == code) \
    printf("Cuda error : %d %s\n", error, #code);

  if(error == hipSuccess) {}
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorMissingConfiguration)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorOutOfMemory)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorNotInitialized)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorLaunchFailure)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorPriorLaunchFailure)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorLaunchTimeOut)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorLaunchOutOfResources)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorInvalidDeviceFunction)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorInvalidConfiguration)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorInvalidDevice)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorInvalidValue)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorInvalidPitchValue)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorInvalidSymbol)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorMapFailed)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorUnmapFailed)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(cudaErrorInvalidHostPointer)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorInvalidDevicePointer)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorInvalidTexture)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(cudaErrorInvalidTextureBinding)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorInvalidChannelDescriptor)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(hipErrorInvalidMemcpyDirection)
  ELSEIF_CUDA_HANDLE_ERROR_CODE(cudaErrorAddressOfConstant)

#undef ELSEIF_HANDLE_ERROR_CODE
}


__global__ void UnconstrainedKernel(float* d_image_vector,
                                    float* d_image_unmixed,
                                    float* d_endmembersInv,
                                    int numSamples,
                                    int num_endmembers,
                                    int num_bands)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (x < numSamples)
    {
    // Unconstrained    
    for(int e = 0; e < num_endmembers; e++)
      {
      d_image_unmixed[e + num_endmembers*x] = 0;
      for (int t = 0; t < num_bands; t++)
        {
        d_image_unmixed[e + num_endmembers*x] += d_endmembersInv[t + e*num_bands] * d_image_vector[ t + num_bands*x ];
        }
      }
    }
}


__global__ void FCLSUKernel(float* d_image_unmixed,int numSamples,
                                    int num_endmembers)
{
  float sum;
  
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < numSamples)
    {
    sum = 0;
    for(int i = 0; i < num_endmembers; i++)
      {
      sum += d_image_unmixed[ i + num_endmembers*x ];
      }
      
    for (int k = 0; k < num_endmembers; k++)
      {
      d_image_unmixed[k + num_endmembers*x] = d_image_unmixed[k + num_endmembers*x ] / sum;
      }
    }
}


__global__ void UnconstrainedISRAKernel(float* d_image_vector,
                                        float* d_image_unmixed,
                                        float* d_image_unmixed_tmp,
                                        float* d_endmembers, 
                                        float* d_endmembersT,
                                        float* d_endmembersInv,
                                        int numSamples,
                                        int num_endmembers,
                                        int num_bands,
                                        int maxiter)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  
  float numerator = 0;
  float denominator = 0;
  float dot = 0;
  
  if (x < numSamples)
    {
    // Unconstrained    
    for(int e = 0; e < num_endmembers; e++)
      {
      d_image_unmixed[e + num_endmembers*x] = 0;
      for (int t = 0; t < num_bands; t++)
        {
        d_image_unmixed[e + num_endmembers*x] += d_endmembersInv[t + e*num_bands] * d_image_vector[ t + num_bands*x ];
        }
      d_image_unmixed_tmp[e + num_endmembers*x] = d_image_unmixed[e + num_endmembers*x];
      }

    // ISRA
    for(int it = 0; it < maxiter; it++)
      {      
      for(int e = 0; e < num_endmembers; e++)
        {
        numerator = 0;
        denominator = 0;
        
        // For all bands
        for (int k = 0; k < num_bands; k++)
          {
//          numerator = numerator + d_endmembers[k + e*num_bands] * l_pixel[k];
          numerator = numerator + d_endmembers[k + e*num_bands] * d_image_vector[ k + num_bands*x ];;
  
          // Calculate dot product
          dot = 0;
          for (int s = 0; s < num_endmembers; s++)
            {
//            dot += d_endmembersT[s + k*num_endmembers] * l_abu[s];
            dot += d_endmembersT[s + k*num_endmembers] * d_image_unmixed_tmp[s + num_endmembers*x];
            }
  
          denominator += dot * d_endmembers[k + e*num_bands];        
          
          }
  
//        l_abu[e] = l_abu[e] * (numerator/denominator);
//        l_abu[e] = numerator/denominator;
        d_image_unmixed[e + num_endmembers*x] = d_image_unmixed_tmp[e + num_endmembers*x] * (numerator/denominator);
        }
        
      for(int e = 0; e < num_endmembers; e++)
        {
        d_image_unmixed_tmp[e + num_endmembers*x] = d_image_unmixed[e + num_endmembers*x];
        }
      }

    }
}

extern "C" void fclsProcessing( float* d_image_vector, 
                                float* d_image_unmixed,
                                float* d_image_unmixed_tmp,
                                float* d_endmembers,
                                float* d_endmembersT,             
                                float* d_endmembersInv,  
                                int numSamples,
                                int numBands,
                                int nbEndmembers,
                                int maxIter,
                                int blockSize)
{
  dim3 dimBlock( blockSize );
  dim3 dimGrid ( (numSamples + blockSize) / blockSize );
  
  printf( " %d \n " , (numSamples + blockSize) / blockSize );
  
  UnconstrainedKernel<<<dimGrid,dimBlock>>>(d_image_vector, d_image_unmixed, d_endmembersInv, numSamples, nbEndmembers, numBands);
  hipDeviceSynchronize();

  //UnconstrainedISRAKernel<<<dimGrid,dimBlock>>>(d_image_vector, d_image_unmixed, d_image_unmixed_tmp, d_endmembers, d_endmembersT, d_endmembersInv, numSamples, nbEndmembers, numBands, maxIter);
  //hipDeviceSynchronize();
  
  //FCLSUKernel<<<dimGrid,dimBlock>>>(d_image_unmixed, numSamples);
  //hipDeviceSynchronize();
}


extern "C" void fclsMallocEndMembers( 
                            float** d_endmembers,
                            float** d_endmembersT,
                            float** d_endmembersInv,
                            int numBands,
                            int nbEndmembers)
{

  float* d_endmembers_ = 0;
  float* d_endmembersT_ = 0;
  float* d_endmembersInv_ = 0;

  printf( "Allocating %d KB\n", numBands*nbEndmembers*sizeof(float) /  1024 );
  CUDA_SAFE_CALL( hipMalloc((void**) &d_endmembers_, nbEndmembers*numBands*sizeof(float)));
  CUDA_SAFE_CALL( hipMemset(d_endmembers_, 0, nbEndmembers*numBands*sizeof(float)));

  printf( "Allocating %d KB\n", numBands*nbEndmembers*sizeof(float) /  1024 );
  CUDA_SAFE_CALL( hipMalloc((void**) &d_endmembersT_, nbEndmembers*numBands*sizeof(float)));
  CUDA_SAFE_CALL( hipMemset(d_endmembersT_, 0, nbEndmembers*numBands*sizeof(float)));

  printf( "Allocating %d KB\n", numBands*nbEndmembers*sizeof(float) /  1024 );
  CUDA_SAFE_CALL( hipMalloc((void**) &d_endmembersInv_, nbEndmembers*numBands*sizeof(float)));
  CUDA_SAFE_CALL( hipMemset(d_endmembersInv_, 0, nbEndmembers*numBands*sizeof(float)));
  
  
  *d_endmembers = d_endmembers_;
  *d_endmembersT = d_endmembersT_;
  *d_endmembersInv = d_endmembersInv_;
}

extern "C" void fclsMallocImage( float** d_image_vector,
                            float** d_image_unmixed,
                            float** d_image_unmixed_tmp,
                            int imageWidth,
                            int imageHeight,
                            int numBands,
                            int nbEndmembers)
{
  printf( "Allocating %d KB\n", numBands*imageWidth*imageHeight*sizeof(float) /  1024);
  CUDA_SAFE_CALL(hipMalloc((void**) d_image_vector, numBands*imageWidth*imageHeight*sizeof(float)));
  CUDA_SAFE_CALL( hipMemset(*d_image_vector, 0, nbEndmembers*numBands*sizeof(float)));

  printf( "Allocating %d KB\n", nbEndmembers*imageWidth*imageHeight*sizeof(float) /  1024);
  CUDA_SAFE_CALL( hipMalloc((void**) d_image_unmixed, nbEndmembers*imageWidth*imageHeight*sizeof(float)));
  CUDA_SAFE_CALL( hipMemset(*d_image_unmixed, 0, nbEndmembers*numBands*sizeof(float)));

  printf( "Allocating %d KB\n", nbEndmembers*imageWidth*imageHeight*sizeof(float) /  1024);
  CUDA_SAFE_CALL( hipMalloc((void**) d_image_unmixed_tmp, nbEndmembers*imageWidth*imageHeight*sizeof(float)));
  CUDA_SAFE_CALL( hipMemset(*d_image_unmixed_tmp, 0, nbEndmembers*numBands*sizeof(float)));
}

         

extern "C" void fclsCopyHostToDevice( float* d_ptr,
                                      const float* h_ptr,
                                      int nb_bytes)
{
  CUDA_SAFE_CALL( hipMemcpy(d_ptr, h_ptr, nb_bytes, hipMemcpyHostToDevice));
}

extern "C" void fclsCopyDeviceToHost( float* h_ptr,
                                      const float* d_ptr,
                                      int nb_bytes)
{
  CUDA_SAFE_CALL( hipMemcpy(h_ptr, d_ptr, nb_bytes, hipMemcpyDeviceToHost));
}

extern "C" void fclsFree( float* d_ptr)
{
  CUDA_SAFE_CALL( hipFree(d_ptr) );
}

extern "C" void fclsInit( void )
{
  if (hipInit(0) != hipSuccess)
    exit (0);
}

